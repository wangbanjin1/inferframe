#include "hip/hip_runtime.h"
//
// Created by fss on 22-12-12.
//
#include "hardswish.hpp"
#include "layer/abstract/layer_factory.hpp"

namespace kuiper_infer {

__global__ void HardswishForward(const int n, const float* in, float* out) {
  CUDA_KERNEL_LOOP(index, n) {
    float val = in[index];
    float result = 0.f;
    if (val <= -3.f) {
      result = 0.f;
    } else if (val >= 3.f) {
      result = val;
    } else {
      result = val * (val + 3) / 6;
    }
    out[index] = result;
  }
}

HardSwishLayer::HardSwishLayer() : Layer("HardSwish") {}

InferStatus HardSwishLayer::Forward(
    const std::vector<std::shared_ptr<Tensor<float>>>& inputs,
    std::vector<std::shared_ptr<Tensor<float>>>& outputs) {
  if (inputs.empty()) {
    LOG(ERROR) << "The input tensor array in the hardswish layer is empty";
    return InferStatus::kInferFailedInputEmpty;
  }

  if (inputs.size() != outputs.size()) {
    LOG(ERROR) << "The input and output tensor array size of the hardswish "
                  "layer do not match";
    return InferStatus::kInferFailedInputOutSizeMatchError;
  }

  if (outputs.empty()) {
    std::shared_ptr<Tensor<float>> output =
        std::make_shared<Tensor<float>>(inputs.at(0)->shapes());
    outputs.push_back(output);
  } else {
    CHECK(inputs.at(0)->shapes() == outputs.at(0)->shapes())
        << "The input tensor and output tensor in the relu layer are not the "
           "same shape";
  }

  uint32_t count = inputs.at(0)->size();
  HardswishForward<<<KUIPER_GET_BLOCKS(count), KUIPER_CUDA_NUM_THREADS>>>(
      count, inputs.at(0)->gpu_data(), outputs.at(0)->gpu_data());
  return InferStatus::kInferSuccess;
  
}

ParseParameterAttrStatus HardSwishLayer::GetInstance(
    const std::shared_ptr<RuntimeOperator>& op,
    std::shared_ptr<Layer>& hardswish_layer) {
  CHECK(op != nullptr) << "HardSwishLayer operator is nullptr";
  hardswish_layer = std::make_shared<HardSwishLayer>();
  return ParseParameterAttrStatus::kParameterAttrParseSuccess;
}

LayerRegistererWrapper kHardSwishGetInstance("nn.Hardswish",
                                             HardSwishLayer::GetInstance);

}  // namespace kuiper_infer